

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_2e_tmpB_cs_kernel(int n_grid1, int n_mo, int ne_b, 
                                     double * wr1, double * mos_l_in_r, double * int2_grad1_u12,
                                     double * tmpB) {


    int i_grid1;
    int ie;
    int p_mo;

    int ix, iy, iz;
    int jx, jjx;

    int n1;
    int n2;

    double wr1_tmp;
    double mol_tmp;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    n1 = 3 * n_grid1;
    n2 = n1 * n_mo;

    while(i_grid1 < n_grid1) {

        wr1_tmp = wr1[i_grid1];

        for(p_mo = 0; p_mo < n_mo; p_mo++) {

            ix = i_grid1 + p_mo * n1;
            iy = ix + n_grid1;
            iz = iy + n_grid1;

            tmpB[ix] = 0.0;
            tmpB[iy] = 0.0;
            tmpB[iz] = 0.0;

            jx = i_grid1 + p_mo * n2;

            for(ie = 0; ie < ne_b; ie++) {

                mol_tmp = mos_l_in_r[i_grid1 + ie * n_grid1];

                jjx = jx + ie * n1;

                tmpB[ix] += wr1_tmp * mol_tmp * int2_grad1_u12[jjx              ];
                tmpB[iy] += wr1_tmp * mol_tmp * int2_grad1_u12[jjx +     n_grid1];
                tmpB[iz] += wr1_tmp * mol_tmp * int2_grad1_u12[jjx + 2 * n_grid1];

            }
        }

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_2e_tmpB_cs(int n_grid1, int n_mo, int ne_b,
                              double * wr1, double * mos_l_in_r, double * int2_grad1_u12,
                              double * tmpB) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_2e_tmpB_cs_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_2e_tmpB_cs_kernel<<<nBlocks, blockSize>>>(n_grid1, n_mo, ne_b,
                                                 wr1, mos_l_in_r, int2_grad1_u12,
                                                 tmpB);

}


