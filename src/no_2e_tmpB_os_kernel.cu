

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_2e_tmpB_os_kernel(int n_grid1, int n_mo, int ne_b, int ne_a,
                                     double * wr1, double * mos_r_in_r, double * int2_grad1_u12,
                                     double * tmpB) {


    int i_grid1;
    int ie;
    int p_mo;

    int ix, iy, iz;
    int jx, jjx;

    int n1;
    int n2;

    double wr1_tmp;
    double mor_tmp;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    n1 = 3 * n_grid1;
    n2 = n1 * n_mo;

    while(i_grid1 < n_grid1) {

        wr1_tmp = wr1[i_grid1];

        for(p_mo = 0; p_mo < n_mo; p_mo++) {

            ix = i_grid1 + p_mo * n1;
            iy = ix + n_grid1;
            iz = iy + n_grid1;

            tmpB[ix] = 0.0;
            tmpB[iy] = 0.0;
            tmpB[iz] = 0.0;

            jx = i_grid1 + p_mo * n2;

            for(ie = 0; ie < ne_b; ie++) {

                mor_tmp = mos_r_in_r[i_grid1 + ie * n_grid1];

                jjx = jx + ie * n1;

                tmpB[ix] += wr1_tmp * mor_tmp * int2_grad1_u12[jjx            ];
                tmpB[iy] += wr1_tmp * mor_tmp * int2_grad1_u12[jjx +   n_grid1];
                tmpB[iz] += wr1_tmp * mor_tmp * int2_grad1_u12[jjx + 2*n_grid1];

            }

            for(ie = ne_b; ie < ne_a; ie++) {

                mor_tmp = mos_r_in_r[i_grid1 + ie * n_grid1];

                jjx = jx + ie * n1;

                tmpB[ix] += 0.5 * wr1_tmp * mor_tmp * int2_grad1_u12[jjx            ];
                tmpB[iy] += 0.5 * wr1_tmp * mor_tmp * int2_grad1_u12[jjx +   n_grid1];
                tmpB[iz] += 0.5 * wr1_tmp * mor_tmp * int2_grad1_u12[jjx + 2*n_grid1];

            }

        }

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_2e_tmpB_os(int n_grid1, int n_mo, int ne_b, int ne_a,
                              double * wr1, double * mos_r_in_r, double * int2_grad1_u12,
                              double * tmpB) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_2e_tmpB_os_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_2e_tmpB_os_kernel<<<nBlocks, blockSize>>>(n_grid1, n_mo, ne_b, ne_a,
                                                 wr1, mos_r_in_r, int2_grad1_u12,
                                                 tmpB);

}


