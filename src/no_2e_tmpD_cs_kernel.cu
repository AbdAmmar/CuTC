

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_2e_tmpD_cs_kernel(int n_grid1, int n_mo,
                                     double * wr1, double * mos_l_in_r, double * mos_r_in_r, double * int2_grad1_u12,
                                     double * tmpD) {


    int i_grid1;
    int p_mo;
    int s_mo;

    int ix, iix;
    int jx, jjx;

    int n1, n2;
    int m1, m2;

    double wr1_tmp;
    double mol_tmp;
    double mor_tmp;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    n1 = 3 * n_grid1;
    n2 = n1 * n_mo;

    m1 = n1 + n_grid1;
    m2 = m1 * n_mo;

    while(i_grid1 < n_grid1) {

        wr1_tmp = wr1[i_grid1];

        for(p_mo = 0; p_mo < n_mo; p_mo++) {

            mol_tmp = mos_l_in_r[i_grid1 + p_mo*n_grid1];

            ix = i_grid1 + p_mo * m1;
            jx = i_grid1 + p_mo * n1;

            for(s_mo = 0; s_mo < n_mo; s_mo++) {

                iix = ix + s_mo * m2;

                jjx = jx + s_mo * n2;

                mor_tmp = mos_r_in_r[i_grid1 + s_mo*n_grid1];

                tmpD[iix            ] = int2_grad1_u12[jjx            ];
                tmpD[iix +   n_grid1] = int2_grad1_u12[jjx +   n_grid1];
                tmpD[iix + 2*n_grid1] = int2_grad1_u12[jjx + 2*n_grid1];
                tmpD[iix + 3*n_grid1] = wr1_tmp * mol_tmp * mor_tmp;

            } // s_mo

        } // p_mo

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_2e_tmpD_cs(int n_grid1, int n_mo,
                              double * wr1, double * mos_l_in_r, double * mos_r_in_r, double * int2_grad1_u12,
                              double * tmpD) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_2e_tmpD_cs_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_2e_tmpD_cs_kernel<<<nBlocks, blockSize>>>(n_grid1, n_mo,
                                                 wr1, mos_l_in_r, mos_r_in_r, int2_grad1_u12,
                                                 tmpD);

}


