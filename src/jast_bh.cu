
#include <hip/hip_runtime.h>


///* compute a**b, where b in [0,10] */
__device__ double powd_int(double a, int b) {

    double r;

    double a2, a3, a4;

    if (b == 0) {
        r = 1.0;
    } else if (b == 1) {
        r = a;
    } else if (b == 2) {
        r = a * a;
    } else if (b == 3) {
        r = a * a * a;
    } else if (b == 4) {
        a2 = a * a;
        r = a2 * a2;
    } else if (b == 5) {
        a2 = a * a;
        r = a * a2 * a2;
    } else if (b == 6) {
        a2 = a * a;
        r = a2 * a2 * a2;
    } else if (b == 7) {
        a2 = a * a;
        a3 = a2 * a;
        r = a2 * a2 * a3;
    } else if (b == 8) {
        a2 = a * a;
        a4 = a2 * a2;
        r = a4 * a4;
    } else if (b == 9) {
        a3 = a * a * a;
        r = a3 * a3 * a3;
    } else if (b == 10) {
        a2 = a * a;
        a4 = a2 * a2;
        r = a2 * a4 * a4;
    }

    return r;
}


__global__ void tc_int_bh_kernel(int ii0, int n_grid1_eff, int n_grid1_tot,
                                 int n_grid1, int n_grid2, int n_nuc, int size_bh,
                                 double *r1, double *r2, double *rn, 
                                 double *c_bh, int *m_bh, int *n_bh, int *o_bh,
                                 double *grad1_u12) {

    /*
        grad1_u12[1] =      [grad1 u(r1,r2)]_x1
        grad1_u12[2] =      [grad1 u(r1,r2)]_y1
        grad1_u12[3] =      [grad1 u(r1,r2)]_z1
        grad1_u12[4] = -0.5 [grad1 u(r1,r2)]^2
    */


//    extern __shared__ char shared_data[];
//    double *shared_c = (double*) shared_data; 
//    int *shared_m = (int*) (shared_data + n_nuc * size_bh * sizeof(double)); 
//    int *shared_n = (int*) (shared_data + n_nuc * size_bh * (sizeof(double)+sizeof(int))); 
//    int *shared_o = (int*) (shared_data + n_nuc * size_bh * (sizeof(double)+2*sizeof(int))); 

    int i_grid1, i_grid2;
    int ii_grid1, ii_grid2, ii_nuc, ii_12;
    int i_nuc;
    int i_bh;
    int ii;
    int jj;
    int kk;

    int m, n, o;
    double c;

    double dx, dy, dz, dist;
    double r1_x, r1_y, r1_z;
    double r2_x, r2_y, r2_z;
    double rn_x, rn_y, rn_z;

    double g12, g12x, g12y, g12z;
    double f1n, f1nx, f1ny, f1nz;
    double f2n;
    double f1n_mm1, f1n_m, f1n_nm1, f1n_n;
    double f2n_m, f2n_n;
    double g12_om1, g12_o;
    double tmp1, tmp2;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

//    i_grid2 = blockIdx.y * blockDim.y + threadIdx.y;
//    if((i_grid1 < n_grid1_eff) && (i_grid2 < n_grid2)) {
//        for(i_nuc = 0; i_nuc < n_nuc; i_nuc++) {
//            ii_nuc = size_bh * i_nuc;
//            for(i_bh = 0; i_bh < size_bh; i_bh++) {
//                kk = i_bh + ii_nuc;
//                shared_c[kk] = c_bh[kk];
//                shared_m[kk] = m_bh[kk];
//                shared_n[kk] = n_bh[kk];
//                shared_o[kk] = o_bh[kk];
//            }
//        }
//    }
//    __syncthreads();


    ii_12 = n_grid1_tot * n_grid2;

    while(i_grid1 < n_grid1_eff) {

        ii = 3 * (ii0 + i_grid1);
        r1_x = r1[ii    ];
        r1_y = r1[ii + 1];
        r1_z = r1[ii + 2];

        ii_grid1 = i_grid1 * n_grid2;

        i_grid2 = blockIdx.y * blockDim.y + threadIdx.y;
        while(i_grid2 < n_grid2) {

            ii_grid2 = ii_grid1 + i_grid2;

            grad1_u12[ii_grid2          ] = 0.0;
            grad1_u12[ii_grid2 +   ii_12] = 0.0;
            grad1_u12[ii_grid2 + 2*ii_12] = 0.0;

            jj = 3 * i_grid2;
            r2_x = r2[jj    ];
            r2_y = r2[jj + 1];
            r2_z = r2[jj + 2];

            // e1-e2 term
            dx = r1_x - r2_x;
            dy = r1_y - r2_y;
            dz = r1_z - r2_z;
            dist = dx * dx + dy * dy + dz * dz;
            if(dist > 1e-15) {
                dist = sqrt(dist);
                tmp1 = 1.0 / (1.0 + dist);
                g12  = dist * tmp1;
                tmp2 = tmp1 * tmp1 / dist;
                g12x = tmp2 * dx;
                g12y = tmp2 * dy;
                g12z = tmp2 * dz;
            } else {
                g12  = 0.0;
                g12x = 0.0;
                g12y = 0.0;
                g12z = 0.0;
            }
            
            for(i_nuc = 0; i_nuc < n_nuc; i_nuc++) {

                rn_x = rn[3*i_nuc  ];
                rn_y = rn[3*i_nuc+1];
                rn_z = rn[3*i_nuc+2];

                // e1-n term
                dx = r1_x - rn_x;
                dy = r1_y - rn_y;
                dz = r1_z - rn_z;
                dist = dx * dx + dy * dy + dz * dz;
                if(dist > 1e-15) {
                    dist = sqrt(dist);
                    tmp1 = 1.0 / (1.0 + dist);
                    f1n  = dist * tmp1;
                    tmp2 = tmp1 * tmp1 / dist;
                    f1nx = tmp2 * dx;
                    f1ny = tmp2 * dy;
                    f1nz = tmp2 * dz;
                } else {
                    f1n  = 0.0;
                    f1nx = 0.0;
                    f1ny = 0.0;
                    f1nz = 0.0;
                }

                // e2-n term
                dx = r2_x - rn_x;
                dy = r2_y - rn_y;
                dz = r2_z - rn_z;
                dist = dx * dx + dy * dy + dz * dz;
                if(dist > 1e-15) {
                    dist = sqrt(dist);
                    f2n  = dist / (1.0 + dist);
                } else {
                    f2n  = 0.0;
                }

                ii_nuc = size_bh * i_nuc;

                for(i_bh = 0; i_bh < size_bh; i_bh++) {

                    kk = i_bh + ii_nuc;

                    //c = shared_c[kk];
                    //if(fabs(c) < 1e-10)
                    //    continue;
                    //m = shared_m[kk];
                    //n = shared_n[kk];
                    //o = shared_o[kk];

                    c = c_bh[kk];
                    if(fabs(c) < 1e-10)
                        continue;
                    m = m_bh[kk];
                    n = n_bh[kk];
                    o = o_bh[kk];

                    // TODO remove
                    if(m == n)
                        c *= 0.5;

                    f1n_m = 1.0;
                    f2n_m = 1.0;
                    if(m > 0) {
                        f1n_mm1 = powd_int(f1n, m-1);
                        f1n_m = f1n_mm1 * f1n;
                        f2n_m = powd_int(f2n, m);
                    }

                    f1n_n = 1.0;
                    f2n_n = 1.0;
                    if(n > 0) {
                        f1n_nm1 = powd_int(f1n, n-1);
                        f1n_n = f1n_nm1 * f1n;
                        f2n_n = powd_int(f2n, n);
                    }

                    tmp1 = 0.0;
                    tmp2 = 0.0;

                    if(m > 0)
                        tmp1 += __int2double_rn(m) * f1n_mm1 * f2n_n;
                    if(n > 0)
                        tmp1 += __int2double_rn(n) * f1n_nm1 * f2n_m;

                    if(o > 0) {

                        g12_om1 = powd_int(g12, o-1);
                        g12_o = g12_om1 * g12;

                        tmp2 = c * __int2double_rn(o) * g12_om1 * (f1n_m * f2n_n + f1n_n * f2n_m);
                        tmp1 = c * tmp1 * g12_o;
                    } else {
                        tmp1 *= c;
                    }

                    grad1_u12[ii_grid2          ] += tmp1 * f1nx + tmp2 * g12x;
                    grad1_u12[ii_grid2 +   ii_12] += tmp1 * f1ny + tmp2 * g12y;
                    grad1_u12[ii_grid2 + 2*ii_12] += tmp1 * f1nz + tmp2 * g12z;

                } // i_bh

            } // i_nuc

            grad1_u12[ii_grid2 + 3*ii_12] = -0.5 * ( grad1_u12[ii_grid2          ] * grad1_u12[ii_grid2          ]
                                                   + grad1_u12[ii_grid2 +   ii_12] * grad1_u12[ii_grid2 +   ii_12]
                                                   + grad1_u12[ii_grid2 + 2*ii_12] * grad1_u12[ii_grid2 + 2*ii_12] );

            i_grid2 += blockDim.y * gridDim.y;

        } // i_grid2

        i_grid1 += blockDim.x * gridDim.x;

    } // i_grid1

}



extern "C" void tc_int_bh(dim3 dimGrid, dim3 dimBlock,
                          int ii0, int n_grid1_eff, int n_grid1_tot,
                          int n_grid1, int n_grid2, int n_nuc, int size_bh,
                          double *r1, double *r2, double *rn, 
                          double *c_bh, int *m_bh, int *n_bh, int *o_bh,
                          double *grad1_u12) {

    //size_t size_sh;
    //size_sh = n_nuc * size_bh * (sizeof(double) + 3 * sizeof(int));
    //tc_int_bh_kernel<<<dimGrid, dimBlock, size_sh>>>(ii0, n_grid1_eff, n_grid1_tot,
    tc_int_bh_kernel<<<dimGrid, dimBlock>>>(ii0, n_grid1_eff, n_grid1_tot,
                                                     n_grid1, n_grid2, n_nuc, size_bh,
                                                     r1, r2, rn, 
                                                     c_bh, m_bh, n_bh, o_bh,
                                                     grad1_u12);
               
}


