
#include <hip/hip_runtime.h>


__global__ void tc_int_bh_kernel(int n_grid1, int n_grid2, int n_nuc, int size_bh,
                                 double *r1, double *r2, double *rn,
                                 double *c_bh, int *m_bh, int *n_bh, int *o_bh,
                                 double *grad1_u12) {

    /*
        grad1_u12[1] =      [grad1 u(r1,r2)]_x1
        grad1_u12[2] =      [grad1 u(r1,r2)]_y1
        grad1_u12[3] =      [grad1 u(r1,r2)]_z1
        grad1_u12[4] = -0.5 [grad1 u(r1,r2)]^2
    */

    int i_grid1, i_grid2;
    int i_nuc;
    int i_bh;
    int i;

    int m, n, o;
    double c;

    double dx, dy, dz, dist;
    double r1_x, r1_y, r1_z;
    double r2_x, r2_y, r2_z;
    double rn_x, rn_y, rn_z;

    double g12, g12x, g12y, g12z;
    double f1n, f1nx, f1ny, f1nz;
    double f2n;
    double f1n_mm1, f1n_m, f1n_nm1, f1n_n;
    double f2n_m, f2n_n;
    double g12_om1, g12_o;
    double tmp1, tmp2;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x ;

    if(i_grid1 < n_grid1) {

        r1_x = r1[i_grid1          ];
        r1_y = r1[i_grid1+  n_grid1];
        r1_z = r1[i_grid1+2*n_grid1];

        for(i_grid2 = 0; i_grid2 < n_grid2; i_grid2++) {

            grad1_u12[i_grid1*n_grid1+i_grid2          ] = 0.0;
            grad1_u12[i_grid1*n_grid1+i_grid2+  n_grid2] = 0.0;
            grad1_u12[i_grid1*n_grid1+i_grid2+2*n_grid2] = 0.0;

            r2_x = r2[i_grid2          ];
            r2_y = r2[i_grid2+  n_grid2];
            r2_z = r2[i_grid2+2*n_grid2];

            // e1-e2 term
            dx = r1_x - r2_x;
            dy = r1_y - r2_y;
            dz = r1_z - r2_z;
            dist = dx * dx + dy * dy + dz * dz;
            if(dist < 1e-15) {
                dist = sqrt(dist);
                tmp1 = 1.0 / (1.0 + dist);
                g12  = dist * tmp1;
                tmp2 = tmp1 * tmp1 / dist;
                g12x = tmp2 * dx;
                g12y = tmp2 * dy;
                g12z = tmp2 * dz;
            } else {
                g12  = 0.0;
                g12x = 0.0;
                g12y = 0.0;
                g12z = 0.0;
            }
            
            for(i_nuc = 0; i_nuc < n_nuc; i_nuc++) {

                rn_x = rn[i_nuc        ];
                rn_y = rn[i_nuc+  n_nuc];
                rn_z = rn[i_nuc+2*n_nuc];

                // e1-n term
                dx = r1_x - rn_x;
                dy = r1_y - rn_y;
                dz = r1_z - rn_z;
                dist = dx * dx + dy * dy + dz * dz;
                if(dist < 1e-15) {
                    dist = sqrt(dist);
                    tmp1 = 1.0 / (1.0 + dist);
                    f1n  = dist * tmp1;
                    tmp2 = tmp1 * tmp1 / dist;
                    f1nx = tmp2 * dx;
                    f1ny = tmp2 * dy;
                    f1nz = tmp2 * dz;
                } else {
                    f1n  = 0.0;
                    f1nx = 0.0;
                    f1ny = 0.0;
                    f1nz = 0.0;
                }

                // e2-n term
                dx = r2_x - rn_x;
                dy = r2_y - rn_y;
                dz = r2_z - rn_z;
                dist = dx * dx + dy * dy + dz * dz;
                if(dist < 1e-15) {
                    dist = sqrt(dist);
                    f2n  = dist / (1.0 + dist);
                } else {
                    f2n  = 0.0;
                }

                for(i_bh = 0; i_bh < size_bh; i_bh++) {

                    c = c_bh[i_bh + size_bh*i_nuc];
                    if(fabs(c) < 1e-10)
                        break;

                    m = m_bh[i_bh + size_bh*i_nuc];
                    n = n_bh[i_bh + size_bh*i_nuc];
                    o = o_bh[i_bh + size_bh*i_nuc];

                    // TODO remove
                    if(m == n)
                        c *= 0.5;

                    f1n_m = 1.0;
                    f2n_m = 1.0;
                    if(m > 0) {
                        f1n_mm1 = 1.0;
                        for(i = 0; i < m-1; i++) {
                            f1n_mm1 *= f1n;
                            f2n_m   *= f2n;
                        }
                        f1n_m = f1n_mm1 * f1n;
                        f2n_m = f2n_m   * f2n;
                    }

                    f1n_n = 1.0;
                    f2n_n = 1.0;
                    if(n > 0) {
                        f1n_nm1 = 1.0;
                        for(i = 0; i < n-1; i++) {
                            f1n_nm1 *= f1n;
                            f2n_n   *= f2n;
                        }
                        f1n_n = f1n_nm1 * f1n;
                        f2n_n = f2n_n   * f2n;
                    }

                    tmp1 = 0.0;
                    tmp2 = 0.0;

                    if(m > 0)
                        tmp1 += __int2double_rn(m) * f1n_mm1 * f2n_n;
                    if(n > 0)
                        tmp1 += __int2double_rn(n) * f1n_nm1 * f2n_m;

                    if(o > 0) {
                        g12_om1 = 1.0;
                        for(i = 0; i < o-1; i++) {
                            g12_om1 *= g12;
                        }
                        g12_o = g12_om1 * g12;

                        tmp2 = c * __int2double_rn(o) * g12_om1 * (f1n_m * f2n_n + f1n_n * f2n_m);
                        tmp1 = c * tmp1 * g12_o;
                    } else {
                        tmp1 *= c;
                    }

                    grad1_u12[i_grid1*n_grid1+i_grid2          ] += tmp1 * f1nx + tmp2 * g12x;
                    grad1_u12[i_grid1*n_grid1+i_grid2+  n_grid2] += tmp1 * f1ny + tmp2 * g12y;
                    grad1_u12[i_grid1*n_grid1+i_grid2+2*n_grid2] += tmp1 * f1nz + tmp2 * g12z;

                } // i_bh

            } // i_nuc

        } // i_grid2

        for(i_grid2 = 0; i_grid2 < n_grid2; i_grid2++) {
            grad1_u12[i_grid1*n_grid1+i_grid2+3*n_grid2] = -0.5 * ( grad1_u12[i_grid1*n_grid1+i_grid2          ] * grad1_u12[i_grid1*n_grid1+i_grid2          ] 
                                                                  + grad1_u12[i_grid1*n_grid1+i_grid2+  n_grid2] * grad1_u12[i_grid1*n_grid1+i_grid2+  n_grid2] 
                                                                  + grad1_u12[i_grid1*n_grid1+i_grid2+2*n_grid2] * grad1_u12[i_grid1*n_grid1+i_grid2+2*n_grid2] ) ;
        }

    } // i_grid1

}


// int tc_int_bh(void) {

    //int n_grid1, n_grid2; 
    ////int ao_num;
    //int n_nuc;
    //int size_bh;

    //int *h_m_bh, *h_n_bh, *h_o_bh;
    //double *h_c_bh; 

    //double *h_r1, *h_r2, *h_rn;
    ////double *h_aos_data1, *h_aos_data2;

    ////double *h_int2_grad1_u12;
    ////double *h_tc_int_2e_ao;

    //int i, j;

    //// ao_num  = 50;
    //n_grid1 = 1000;
    //n_grid2 = 10000;
    //n_nuc = 5;
    //size_bh = 10;

    //h_r1 = (double*) malloc(size_r1);
    //h_r2 = (double*) malloc(size_r2);
    //h_rn = (double*) malloc(size_rn);

    //h_c_bh = (double*) malloc(size_jbh1);
    //h_m_bh = (int*) malloc(size_jbh2);
    //h_n_bh = (int*) malloc(size_jbh2);
    //h_o_bh = (int*) malloc(size_jbh2);

    //for(i = 0; i < n_grid1; i++) {
    //    h_r1[i          ] = 0.1;
    //    h_r1[i+  n_grid1] = 0.1;
    //    h_r1[i+2*n_grid1] = 0.1;
    //}
    //for(i = 0; i < n_grid2; i++) {
    //    h_r2[i          ] = 0.2;
    //    h_r2[i+  n_grid2] = 0.2;
    //    h_r2[i+2*n_grid2] = 0.2;
    //}
    //for(i = 0; i < n_nuc; i++) {
    //    h_rn[i        ] = 0.3;
    //    h_rn[i+  n_nuc] = 0.3;
    //    h_rn[i+2*n_nuc] = 0.3;
    //}
    //for (j = 0; j < n_nuc; j++) {
    //    for (i = 0; i < size_bh; i++) {
    //        h_c_bh[i + j*n_nuc] = 0.5;
    //        h_m_bh[i + j*n_nuc] = 2;
    //        h_n_bh[i + j*n_nuc] = 3;
    //        h_o_bh[i + j*n_nuc] = 4;
    //    }
    //}


extern "C" void tc_int_bh(int n_grid1, int n_grid2, int ao_num, int n_nuc, int size_bh,
                          int *h_m_bh, int *h_n_bh, int *h_o_bh, double *h_c_bh,
                          double *h_r1, double *h_r2, double *h_rn,
                          double *h_aos_data1, double *h_aos_data2,
                          double *h_int2_grad1_u12, double *h_tc_int_2e_ao) {

    int *d_m_bh, *d_n_bh, *d_o_bh;
    double *d_c_bh; 

    double *d_r1, *d_r2, *d_rn;

    //double *d_aos_data1, *d_aos_data2;


    double *d_grad1_u12;
    //double *d_int2_grad1_u12;
    //double *d_tc_int_2e_ao;

    size_t size_r1, size_r2, size_rn;
    //size_t size_aos_r1, size_aos_r2;
    size_t size_r12;
    //size_t size_int1, size_int2;
    size_t size_jbh1, size_jbh2;

    int threadsPerBlock, numBlocks;

    size_r1 = 3 * n_grid1 * sizeof(double);
    size_r2 = 3 * n_grid2 * sizeof(double);
    size_rn = 3 * n_nuc   * sizeof(double);

    size_r12 = 4 * n_grid1 * n_grid2 * sizeof(double);

    //size_aos_r1 = 4 * n_grid1 * ao_num * sizeof(double);
    //size_aos_r2 = 4 * n_grid2 * ao_num * sizeof(double);

    //size_int1 = 4 * n_grid2 * ao_num * ao_num * sizeof(double);
    //size_int2 = ao_num * ao_num * ao_num * ao_num * sizeof(double);

    size_jbh1 = size_bh * sizeof(double);
    size_jbh2 = size_bh * sizeof(int);


    hipMalloc(&d_r1, size_r1);
    hipMalloc(&d_r2, size_r2);
    hipMalloc(&d_rn, size_rn);

    // cudaMalloc(&d_aos_data1, size_aos_r1);
    // cudaMalloc(&d_aos_data2, size_aos_r2);

    hipMalloc(&d_grad1_u12, size_r12);

    // cudaMalloc(&d_int2_grad1_u12, size_int1);
    // cudaMalloc(&d_tc_int_2e_ao, size_int2);

    hipMalloc(&d_c_bh, size_jbh1);
    hipMalloc(&d_m_bh, size_jbh2);
    hipMalloc(&d_n_bh, size_jbh2);
    hipMalloc(&d_o_bh, size_jbh2);

    hipMemcpy(d_r1, h_r1, size_r1, hipMemcpyHostToDevice);
    hipMemcpy(d_r2, h_r2, size_r2, hipMemcpyHostToDevice);
    hipMemcpy(d_rn, h_rn, size_rn, hipMemcpyHostToDevice);

    // cudaMemcpy(d_aos_data1, h_aos_data1, size_aos_r1, cudaMemcpyHostToDevice);
    // cudaMemcpy(d_aos_data2, h_aos_data2, size_aos_r2, cudaMemcpyHostToDevice);

    hipMemcpy(d_c_bh, h_c_bh, size_jbh1, hipMemcpyHostToDevice);
    hipMemcpy(d_m_bh, h_m_bh, size_jbh2, hipMemcpyHostToDevice);
    hipMemcpy(d_n_bh, h_n_bh, size_jbh2, hipMemcpyHostToDevice);
    hipMemcpy(d_o_bh, h_o_bh, size_jbh2, hipMemcpyHostToDevice);


    threadsPerBlock = 16;
    numBlocks = (n_grid1 + threadsPerBlock - 1) / threadsPerBlock;

    tc_int_bh_kernel<<<numBlocks, threadsPerBlock>>>(n_grid1, n_grid2, n_nuc, size_bh,
                                                     d_r1, d_r2, d_rn,
                                                     d_c_bh, d_m_bh, d_n_bh, d_o_bh,
                                                     d_grad1_u12);



    //cudaMemcpy(h_int2_grad1_u12, d_int2_grad1_u12, size_int1, cudaMemcpyDeviceToHost);

    hipFree(d_r1);
    hipFree(d_r2);
    hipFree(d_rn);

}


