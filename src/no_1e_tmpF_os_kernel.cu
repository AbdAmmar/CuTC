

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_1e_tmpF_os_kernel(int n_grid1, int n_mo, int ne_b, int ne_a,
                                     double * wr1, double * mos_r_in_r, double * int2_grad1_u12,
                                     double * tmpS, double * tmpJ, double * tmpR,
                                     double * tmpF) {


    int i_grid1;
    int ie;
    int je;
    int p_mo;

    int iix;
    int jx, jjx;
    int kx, kkx;
    int lx, llx;

    int iR, iF;

    int n1, n2;
    int m1;

    double wr1_tmp;
    double mor_tmp, mor_i, mor_j;
    double S;
    double Jx, Jy, Jz;
    double Rx, Ry, Rz;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    n1 = 3 * n_grid1;
    n2 = n1 * n_mo;

    m1 = 5 * n_grid1;

    while(i_grid1 < n_grid1) {

        wr1_tmp = wr1[i_grid1];

        S = tmpS[i_grid1];

        Jx = tmpJ[i_grid1            ];
        Jy = tmpJ[i_grid1 +   n_grid1];
        Jz = tmpJ[i_grid1 + 2*n_grid1];

        for(p_mo = 0; p_mo < n_mo; p_mo++) {

            mor_tmp = mos_r_in_r[i_grid1 + p_mo*n_grid1];

            iR = i_grid1 + p_mo * n1;
            iF = i_grid1 + p_mo * m1;

            Rx = tmpR[iR            ];
            Ry = tmpR[iR +   n_grid1];
            Rz = tmpR[iR + 2*n_grid1];

            tmpF[iF            ] = -2.0 * (Rx * Jx + Ry * Jy + Rz * Jz) + mor_tmp * S;
            tmpF[iF +   n_grid1] = wr1_tmp * mor_tmp;
            tmpF[iF + 2*n_grid1] = Rx;
            tmpF[iF + 3*n_grid1] = Ry;
            tmpF[iF + 4*n_grid1] = Rz;

            jx = i_grid1 + p_mo * n2;

            for(ie = 0; ie < ne_b; ie++) {

                mor_i = mos_r_in_r[i_grid1 + ie*n_grid1];

                kx = i_grid1 + ie * n1;

                for(je = 0; je < ne_b; je++) {

                    jjx = jx + je * n1;
                    kkx = kx + je * n2;

                    tmpF[iF] += mor_i * int2_grad1_u12[jjx            ] * int2_grad1_u12[kkx            ];
                    tmpF[iF] += mor_i * int2_grad1_u12[jjx +   n_grid1] * int2_grad1_u12[kkx +   n_grid1];
                    tmpF[iF] += mor_i * int2_grad1_u12[jjx + 2*n_grid1] * int2_grad1_u12[kkx + 2*n_grid1];

                } // ie

            } // je

            for(ie = ne_b; ie < ne_a; ie++) {

                mor_i = mos_r_in_r[i_grid1 + ie*n_grid1];

                kx = i_grid1 + ie * n1;
                lx = i_grid1 + ie * n2;

                iix = jx + ie * n1;

                for(je = 0; je < ne_b; je++) {

                    mor_j = mos_r_in_r[i_grid1 + je*n_grid1];

                    jjx = jx + je * n1;
                    kkx = kx + je * n2;
                    llx = lx + je * n1;

                    tmpF[iF] += 0.5 * (mor_i * int2_grad1_u12[jjx            ] * int2_grad1_u12[kkx            ] + mor_j * int2_grad1_u12[iix            ] * int2_grad1_u12[llx            ]);
                    tmpF[iF] += 0.5 * (mor_i * int2_grad1_u12[jjx +   n_grid1] * int2_grad1_u12[kkx +   n_grid1] + mor_j * int2_grad1_u12[iix +   n_grid1] * int2_grad1_u12[llx +   n_grid1]);
                    tmpF[iF] += 0.5 * (mor_i * int2_grad1_u12[jjx + 2*n_grid1] * int2_grad1_u12[kkx + 2*n_grid1] + mor_j * int2_grad1_u12[iix + 2*n_grid1] * int2_grad1_u12[llx + 2*n_grid1]);

                } // ie

            } // je

            for(ie = ne_b; ie < ne_a; ie++) {

                mor_i = mos_r_in_r[i_grid1 + ie*n_grid1];

                kx = i_grid1 + ie * n1;

                for(je = ne_b; je < ne_a; je++) {

                    jjx = jx + je * n1;
                    kkx = kx + je * n2;

                    tmpF[iF] += 0.5 * mor_i * int2_grad1_u12[jjx            ] * int2_grad1_u12[kkx            ];
                    tmpF[iF] += 0.5 * mor_i * int2_grad1_u12[jjx +   n_grid1] * int2_grad1_u12[kkx +   n_grid1];
                    tmpF[iF] += 0.5 * mor_i * int2_grad1_u12[jjx + 2*n_grid1] * int2_grad1_u12[kkx + 2*n_grid1];

                } // ie

            } // je
    
        } // p_mo

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_1e_tmpF_os(int n_grid1, int n_mo, int ne_b, int ne_a,
                              double * wr1, double * mos_r_in_r, double * int2_grad1_u12,
                              double * tmpS, double * tmpJ, double * tmpR,
                              double * tmpF) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_1e_tmpF_os_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_1e_tmpF_os_kernel<<<nBlocks, blockSize>>>(n_grid1, n_mo, ne_b, ne_a,
                                                 wr1, mos_r_in_r, int2_grad1_u12,
                                                 tmpS, tmpJ, tmpR,
                                                 tmpF);

}


