

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_1e_tmpR_os_kernel(int n_grid1, int n_mo, int ne_b, int ne_a,
                                     double * mos_r_in_r, double * int2_grad1_u12,
                                     double * tmpR) {


    int i_grid1;
    int ie;
    int p_mo;

    int ix, iy, iz;
    int jx, jjx;

    int n1;
    int n2;

    double mor_tmp;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    n1 = 3 * n_grid1;
    n2 = n1 * n_mo;

    while(i_grid1 < n_grid1) {

        for(p_mo = 0; p_mo < n_mo; p_mo++) {

            ix = i_grid1 + p_mo * n1;
            iy = ix + n_grid1;
            iz = iy + n_grid1;

            tmpR[ix] = 0.0;
            tmpR[iy] = 0.0;
            tmpR[iz] = 0.0;

            jx = i_grid1 + p_mo * n2;

            for(ie = 0; ie < ne_b; ie++) {

                mor_tmp = mos_r_in_r[i_grid1 + ie * n_grid1];

                jjx = jx + ie * n1;

                tmpR[ix] += mor_tmp * int2_grad1_u12[jjx            ];
                tmpR[iy] += mor_tmp * int2_grad1_u12[jjx +   n_grid1];
                tmpR[iz] += mor_tmp * int2_grad1_u12[jjx + 2*n_grid1];

            }

            for(ie = ne_b; ie < ne_a; ie++) {

                mor_tmp = mos_r_in_r[i_grid1 + ie * n_grid1];

                jjx = jx + ie * n1;

                tmpR[ix] += 0.5 * mor_tmp * int2_grad1_u12[jjx            ];
                tmpR[iy] += 0.5 * mor_tmp * int2_grad1_u12[jjx +   n_grid1];
                tmpR[iz] += 0.5 * mor_tmp * int2_grad1_u12[jjx + 2*n_grid1];

            }

        }

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_1e_tmpR_os(int n_grid1, int n_mo, int ne_b, int ne_a,
                              double * mos_r_in_r, double * int2_grad1_u12,
                              double * tmpR) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_1e_tmpR_os_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_1e_tmpR_os_kernel<<<nBlocks, blockSize>>>(n_grid1, n_mo, ne_b, ne_a,
                                                 mos_r_in_r, int2_grad1_u12,
                                                 tmpR);

}

