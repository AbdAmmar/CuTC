

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_1e_tmpL_cs_kernel(int n_grid1, int n_mo, int ne_b, 
                                     double * mos_l_in_r, double * int2_grad1_u12,
                                     double * tmpL) {


    int i_grid1;
    int ie;
    int p_mo;

    int ix, iy, iz;
    int iix;

    int n1;
    int n2;

    double mol_tmp;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    n1 = 3 * n_grid1;
    n2 = n1 * n_mo;

    while(i_grid1 < n_grid1) {

        for(p_mo = 0; p_mo < n_mo; p_mo++) {

            ix = i_grid1 + p_mo * n1;
            iy = ix + n_grid1;
            iz = iy + n_grid1;

            tmpL[ix] = 0.0;
            tmpL[iy] = 0.0;
            tmpL[iz] = 0.0;

            for(ie = 0; ie < ne_b; ie++) {

                mol_tmp = mos_l_in_r[i_grid1 + ie * n_grid1];

                iix = ix + ie * n2;

                tmpL[ix] += mol_tmp * int2_grad1_u12[iix            ];
                tmpL[iy] += mol_tmp * int2_grad1_u12[iix +   n_grid1];
                tmpL[iz] += mol_tmp * int2_grad1_u12[iix + 2*n_grid1];

            }
        }

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_1e_tmpL_cs(int n_grid1, int n_mo, int ne_b,
                              double * mos_l_in_r, double * int2_grad1_u12,
                              double * tmpL) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_1e_tmpL_cs_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_1e_tmpL_cs_kernel<<<nBlocks, blockSize>>>(n_grid1, n_mo, ne_b,
                                                 mos_l_in_r, int2_grad1_u12,
                                                 tmpL);

}


