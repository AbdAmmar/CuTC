
#include <stdio.h>
#include <hipblas.h>

#include "short_range_integ_herm.cuh"
#include "short_range_integ_nonherm.cuh"
#include "utils.cuh"
#include "add_trans_inplace.cuh"


extern "C" void get_int_2e_ao(int n_grid1, int n_ao, double *wr1, double *aos_data1,
                              double *int2_grad1_u12, double *int_2e_ao) {


    double *int_fct_short_range_herm;
    double *int_fct_short_range_nonherm;

    double alpha, beta;

    int blockSize = 32;
    int nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching int_short_range_herm_kernel & int_short_range_nonherm_kernel and with %d blocks and %d threads/block\n", 
            nBlocks, blockSize);


    hipblasHandle_t handle;

    checkCublasErrors(hipblasCreate(&handle), "hipblasCreate", __FILE__, __LINE__);


    // Hermitian part

    checkCudaErrors(hipMalloc((void**)&int_fct_short_range_herm, n_grid1 * n_ao * n_ao * sizeof(double)), "hipMalloc", __FILE__, __LINE__);

    int_short_range_herm_kernel<<<nBlocks, blockSize>>>(n_grid1, n_ao, wr1, aos_data1, int_fct_short_range_herm);

    checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
    checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);

    alpha = 1.0;
    beta = 0.0;

    checkCublasErrors( hipblasDgemm( handle
                                  , HIPBLAS_OP_N, HIPBLAS_OP_N
                                  , n_ao*n_ao, n_ao*n_ao, n_grid1
                                  , &alpha
                                  , &int2_grad1_u12[n_ao*n_ao*n_grid1*3], n_ao*n_ao
                                  , &int_fct_short_range_herm[0], n_grid1
                                  , &beta
                                  , &int_2e_ao[0], n_ao*n_ao )
                     , "hipblasDgemm", __FILE__, __LINE__);

    checkCudaErrors(hipFree(int_fct_short_range_herm), "hipFree", __FILE__, __LINE__);

    // // //



    // non-Hermitian part

    checkCudaErrors(hipMalloc((void**)&int_fct_short_range_nonherm, 3*n_grid1*n_ao*n_ao*sizeof(double)), "hipMalloc", __FILE__, __LINE__);

    int_short_range_nonherm_kernel<<<nBlocks, blockSize>>>(n_grid1, n_ao, wr1, aos_data1, int_fct_short_range_nonherm);

    checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
    checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);

    alpha = -0.5;
    beta = 1.0;
    checkCublasErrors( hipblasDgemm( handle
                                  , HIPBLAS_OP_N, HIPBLAS_OP_N
                                  , n_ao*n_ao, n_ao*n_ao, 3*n_grid1
                                  , &alpha
                                  , &int2_grad1_u12[0], n_ao*n_ao
                                  , &int_fct_short_range_nonherm[0], 3*n_grid1
                                  , &beta
                                  , &int_2e_ao[0], n_ao*n_ao )
                     , "hipblasDgemm", __FILE__, __LINE__);

    checkCudaErrors(hipFree(int_fct_short_range_nonherm), "hipFree", __FILE__, __LINE__);

    // // //


    // int_2e_ao <-- int_2e_ao + int_2e_ao.T

    int sBlocks = 32;
    int nbBlocks = (n_ao*n_ao + sBlocks - 1) / sBlocks;

    dim3 dimGrid(nbBlocks, nbBlocks, 1);
    dim3 dimBlock(sBlocks, sBlocks, 1);

    trans_inplace_kernel<<<dimGrid, dimBlock>>>(int_2e_ao, n_ao*n_ao);
    checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);

    // // //

    checkCublasErrors(hipblasDestroy(handle), "hipblasDestroy", __FILE__, __LINE__);

}


