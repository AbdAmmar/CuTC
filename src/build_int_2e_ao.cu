
#include <hipblas.h>

#include "short_range_integ_herm.cuh"
#include "short_range_integ_nonherm.cuh"


extern "C" void get_int_2e_ao(int nBlocks, int blockSize,
                              int n_grid1, int n_ao, double *wr1, double *aos_data1,
                              double *int2_grad1_u12, double *int_2e_ao) {


    double *int_fct_short_range_herm;
    double *int_fct_short_range_nonherm;

    double alpha, beta;

    hipblasHandle_t handle;


    // Hermitian part

    hipMalloc((void**)&int_fct_short_range_herm, n_grid1 * n_ao * n_ao * sizeof(double));

    int_short_range_herm_kernel<<<nBlocks, blockSize>>>(n_grid1, n_ao, wr1, aos_data1, int_fct_short_range_herm);
    hipDeviceSynchronize();

    hipblasCreate(&handle);

    hipblasDgemm( handle
               , HIPBLAS_OP_N, HIPBLAS_OP_N
               , n_ao*n_ao, n_ao*n_ao, n_grid1
               , &alpha
               , &int2_grad1_u12[n_ao*n_ao*n_grid1*3], n_ao*n_ao
               , &int_fct_short_range_herm[0], n_grid1
               , &beta
               , &int_2e_ao[0], n_ao*n_ao );

    hipblasDestroy(handle);

    hipFree(int_fct_short_range_herm);

    // // //



    // non-Hermitian part

    hipMalloc((void**)&int_fct_short_range_nonherm, 3*n_grid1*n_ao*n_ao*sizeof(double));
    int_short_range_nonherm_kernel<<<nBlocks, blockSize>>>(n_grid1, n_ao, wr1, aos_data1, int_fct_short_range_nonherm);

    hipblasCreate(&handle);

    alpha = -1.0;
    beta = 1.0;
    hipblasDgemm( handle
               , HIPBLAS_OP_N, HIPBLAS_OP_N
               , n_ao*n_ao, n_ao*n_ao, 3*n_grid1
               , &alpha
               , &int2_grad1_u12[0], n_ao*n_ao
               , &int_fct_short_range_nonherm[0], 3*n_grid1
               , &beta
               , &int_2e_ao[0], n_ao*n_ao );

    hipblasDestroy(handle);

    hipFree(int_fct_short_range_nonherm);

    // // //


    // int_2e_ao <-- int_2e_ao + int_2e_ao.T

    hipblasCreate(&handle);

    alpha = 1.0;
    beta = 1.0;
    hipblasDgeam( handle
               , HIPBLAS_OP_T, HIPBLAS_OP_N
               , n_ao*n_ao, n_ao*n_ao
               , &alpha
               , &int_2e_ao[0], n_ao*n_ao
               , &beta
               , &int_2e_ao[0], n_ao*n_ao
               , &int_2e_ao[0], n_ao*n_ao );

    hipblasDestroy(handle);

    // // //

}


