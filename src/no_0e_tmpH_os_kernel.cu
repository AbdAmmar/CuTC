

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_0e_tmpH_os_kernel(int n_grid1, int n_mo, int ne_b, int ne_a,
                                     double * mos_r_in_r, double * int2_grad1_u12,
                                     double * tmpH) {


    int i_grid1;
    int ie;
    int je;

    int ix, iy, iz;
    int jx, jjx;

    int n1;
    int n2;

    double mor_tmp;


    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    n1 = 3 * n_grid1;
    n2 = n1 * n_mo;

    while(i_grid1 < n_grid1) {

        for(je = 0; je < ne_b; je++) {

            ix = i_grid1 + je * n1;
            iy = ix + n_grid1;
            iz = iy + n_grid1;

            tmpH[ix] = 0.0;
            tmpH[iy] = 0.0;
            tmpH[iz] = 0.0;

            jx = i_grid1 + je * n2;

            for(ie = ne_b; ie < ne_a; ie++) {

                mor_tmp = mos_r_in_r[i_grid1 + ie * n_grid1];

                jjx = jx + ie * n1;

                tmpH[ix] += 0.5 * mor_tmp * int2_grad1_u12[jjx            ];
                tmpH[iy] += 0.5 * mor_tmp * int2_grad1_u12[jjx +   n_grid1];
                tmpH[iz] += 0.5 * mor_tmp * int2_grad1_u12[jjx + 2*n_grid1];

            }

        }

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_0e_tmpH_os(int n_grid1, int n_mo, int ne_b, int ne_a,
                              double * mos_r_in_r, double * int2_grad1_u12,
                              double * tmpH) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_0e_tmpH_os_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_0e_tmpH_os_kernel<<<nBlocks, blockSize>>>(n_grid1, n_mo, ne_b, ne_a,
                                                 mos_r_in_r, int2_grad1_u12,
                                                 tmpH);

}


