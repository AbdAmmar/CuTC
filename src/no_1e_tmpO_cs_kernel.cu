

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_1e_tmpO_cs_kernel(int n_grid1, int ne_b, 
                                     double * mos_l_in_r, double * mos_r_in_r,
                                     double * tmpO) {


    int i_grid1;
    int ie;
    int i_mo;

    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    while(i_grid1 < n_grid1) {

        tmpO[i_grid1] = 0.0;

        for(ie = 0; ie < ne_b; ie++) {

            i_mo = i_grid1 + ie * n_grid1;

            tmpO[i_grid1] += mos_l_in_r[i_mo] * mos_r_in_r[i_mo];

        }

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_1e_tmpO_cs(int n_grid1, int ne_b,
                              double * mos_l_in_r, double * mos_r_in_r,
                              double * tmpO) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_1e_tmpO_cs_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_1e_tmpO_cs_kernel<<<nBlocks, blockSize>>>(n_grid1, ne_b,
                                                 mos_l_in_r, mos_r_in_r,
                                                 tmpO);

}


