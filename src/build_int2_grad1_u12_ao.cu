#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hipblas.h>

#include "jast_bh.cuh"
#include "long_range_integ.cuh"
#include "utils.cuh"

extern "C" void get_int2_grad1_u12_ao(dim3 dimGrid, dim3 dimBlock,
                                      int n_grid1, int n_grid2, int n_ao, int n_nuc, int size_bh,
                                      double *r1, double *r2, double *wr2, double *rn, double *aos_data2,
                                      double *c_bh, int *m_bh, int *n_bh, int *o_bh, 
                                      double *int2_grad1_u12_ao) {


    int i_pass;
    int ii, jj, kk;
    int jj0, kk0;
    int n_grid1_pass, n_grid1_rest, n_pass;

    double *int_fct_long_range;
    double *grad1_u12;

    int m;
    double alpha, beta;

    size_t size_sh_mem;
    size_t free_mem, total_mem;

    double n_tmp;

    int blockSize = 32;
    int nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching int_long_range_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    size_sh_mem = n_nuc * size_bh * (sizeof(double) + 3 * sizeof(int)) 
                + 3 * n_nuc * sizeof(double);

    hipblasHandle_t myhandle;

    checkCublasErrors(hipblasCreate(&myhandle), "hipblasCreate", __FILE__, __LINE__);

    alpha = 1.0;
    beta = 0.0;

    jj0 = n_ao * n_ao;

    checkCudaErrors(hipMalloc((void**)&int_fct_long_range, n_grid2 * n_ao * n_ao * sizeof(double)), "hipMalloc", __FILE__, __LINE__);

    int_long_range_kernel<<<nBlocks, blockSize>>>(0, n_grid2, n_grid2,
                                                  n_grid2, n_ao, wr2, aos_data2, int_fct_long_range);
    checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
    checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);



    checkCudaErrors(hipMemGetInfo(&free_mem, &total_mem), "hipMemGetInfo", __FILE__, __LINE__);

    n_tmp = (((double)free_mem - 0.5e9) / 8.0) / (4.0 * (double) n_grid2);
    if(n_tmp < 1.0*n_grid1) {
        if(n_tmp > 1.0) {
            n_grid1_pass = (int) n_tmp;
        } else {
            n_grid1_pass = 1;
        }
    } else {
        n_grid1_pass = n_grid1;
    }

    n_grid1_rest = (int) fmod(1.0 * n_grid1, 1.0 * n_grid1_pass);
    n_pass = (int) ((n_grid1 - n_grid1_rest) / n_grid1_pass);

    printf("n_grid1_pass = %d\n", n_grid1_pass);
    printf("n_grid1_rest = %d\n", n_grid1_rest);
    printf("n_pass = %d\n", n_pass);



    checkCudaErrors(hipMalloc((void**)&grad1_u12, 4 * n_grid1_pass * n_grid2 * sizeof(double)), "hipMalloc", __FILE__, __LINE__);

    kk0 = n_grid1_pass * n_grid2;

    for (i_pass = 0; i_pass < n_pass; i_pass++) {

        ii = i_pass * n_grid1_pass;

        // TODO
        tc_int_bh_kernel<<<dimGrid, dimBlock, size_sh_mem>>>(ii, n_grid1_pass, n_grid1_pass,
                                                             0, n_grid2, n_grid2,
                                                             n_nuc, size_bh,
                                                             r1, r2, rn,
                                                             c_bh, m_bh, n_bh, o_bh,
                                                             grad1_u12);
        checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
        checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);
    
        for (m = 0; m < 4; m++) {
            jj = jj0 * (ii + m * n_grid1);
            kk = kk0 * m;
            checkCublasErrors( hipblasDgemm( myhandle
                                          , HIPBLAS_OP_T, HIPBLAS_OP_N
                                          , n_ao*n_ao, n_grid1_pass, n_grid2
                                          , &alpha
                                          , &int_fct_long_range[0], n_grid2
                                          , &grad1_u12[kk], n_grid2
                                          , &beta
                                          , &int2_grad1_u12_ao[jj], n_ao*n_ao )
                             , "hipblasDgemm", __FILE__, __LINE__);
        }

    }
    
    if(n_grid1_rest > 0) {

        ii = n_pass * n_grid1_pass;

        // TODO
        tc_int_bh_kernel<<<dimGrid, dimBlock, size_sh_mem>>>(ii, n_grid1_rest, n_grid1_pass,
                                                             0, n_grid2, n_grid2,
                                                             n_nuc, size_bh,
                                                             r1, r2, rn,
                                                             c_bh, m_bh, n_bh, o_bh,
                                                             grad1_u12);
        checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
        checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);
    
        for (m = 0; m < 4; m++) {
            jj = jj0 * (ii + m * n_grid1);
            kk = kk0 * m;
            checkCublasErrors( hipblasDgemm( myhandle
                                          , HIPBLAS_OP_T, HIPBLAS_OP_N
                                          , n_ao*n_ao, n_grid1_rest, n_grid2
                                          , &alpha
                                          , &int_fct_long_range[0], n_grid2
                                          , &grad1_u12[kk], n_grid2
                                          , &beta
                                          , &int2_grad1_u12_ao[jj], n_ao*n_ao )
                             , "hipblasDgemm", __FILE__, __LINE__);
        }

    }


    checkCublasErrors(hipblasDestroy(myhandle), "hipblasDestroy", __FILE__, __LINE__);

    checkCudaErrors(hipFree(int_fct_long_range), "hipFree", __FILE__, __LINE__);
    checkCudaErrors(hipFree(grad1_u12), "hipFree", __FILE__, __LINE__);

}



