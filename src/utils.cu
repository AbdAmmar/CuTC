
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <hipblas.h>


extern "C" void checkCudaErrors(hipError_t err, const char* msg, const char* file, int line) {
    if (err != hipSuccess) {
        printf("CUDA Error in %s at line %d\n", file, line);
        printf("%s - %s\n", msg, hipGetErrorString(err));
        exit(0);
    }
}


const char* cublasGetErrorString(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "CUBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:
            return "CUBLAS_STATUS_LICENSE_ERROR";
    }
    return "UNKNOWN CUBLAS ERROR";
}

extern "C" void checkCublasErrors(hipblasStatus_t status, const char* msg, const char* file, int line) {

    const char* err = cublasGetErrorString(status);

    if (err != "CUBLAS_STATUS_SUCCESS") {
        printf("CUBLAS Error in %s at line %d\n", file, line);
        printf("%s - %s\n", msg, err);
        exit(0);
    }
}



// Function to check if P2P is supported between two devices
extern "C" int checkPeerToPeerSupport(int device1, int device2) {
    int canAccessPeer = 0;
    hipError_t err;

    // Check if device1 can access device2
    err = hipDeviceCanAccessPeer(&canAccessPeer, device1, device2);
    if (err != hipSuccess) {
        printf("Error checking peer access: %s\n", hipGetErrorString(err));
        return 0;
    }

    return canAccessPeer;
}

