
#include <hip/hip_runtime.h>

#include <stdio.h>



extern "C" void checkCudaErrors(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        printf("CUDA Error: %f - %f", msg, hipGetErrorString(err));
        exit(0);
    }
}

