
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <hipblas.h>



extern "C" void checkCudaErrors(hipError_t err, const char* msg, const char* file, int line) {
    if (err != hipSuccess) {
        printf("CUDA Error in %s at line %d\n", file, line);
        printf("%s - %s\n", msg, hipGetErrorString(err));
        exit(0);
    }
}


const char* cublasGetErrorString(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "CUBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:
            return "CUBLAS_STATUS_LICENSE_ERROR";
    }
    return "UNKNOWN CUBLAS ERROR";
}

extern "C" void checkCublasErrors(hipblasStatus_t status, const char* msg, const char* file, int line) {

    const char* err = cublasGetErrorString(status);

    if (err != "CUBLAS_STATUS_SUCCESS") {
        printf("CUBLAS Error in %s at line %d\n", file, line);
        printf("%s - %s\n", msg, err);
        exit(0);
    }
}


