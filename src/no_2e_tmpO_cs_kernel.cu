

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_2e_tmpO_cs_kernel(int n_grid1, int ne_b, 
                                     double * wr1, double * mos_l_in_r, double * mos_r_in_r,
                                     double * tmpO) {


    int i_grid1;
    int ie;
    int i_mo;

    double wr1_tmp;

    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    while(i_grid1 < n_grid1) {

        tmpO[i_grid1] = 0.0;

        wr1_tmp = wr1[i_grid1];

        for(ie = 0; ie < ne_b; ie++) {

            i_mo = i_grid1 + ie * n_grid1;

            tmpO[i_grid1] += wr1_tmp * mos_l_in_r[i_mo] * mos_r_in_r[i_mo];

        }

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_2e_tmpO_cs(int n_grid1, int ne_b,
                              double * wr1, double * mos_l_in_r, double * mos_r_in_r,
                              double * tmpO) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_2e_tmpO_cs_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_2e_tmpO_cs_kernel<<<nBlocks, blockSize>>>(n_grid1, ne_b,
                                                 wr1, mos_l_in_r, mos_r_in_r,
                                                 tmpO);

}

