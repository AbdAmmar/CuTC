

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void no_2e_tmpJ_cs_kernel(int n_grid1, int n_mo, int ne_b, 
                                     double * wr1, double * int2_grad1_u12,
                                     double * tmpJ) {


    int i_grid1;
    int ie;

    int ii_grid1;

    int nn;

    double wr1_tmp;

    i_grid1 = blockIdx.x * blockDim.x + threadIdx.x;

    nn = 3 * n_grid1 * (1 + n_mo);

    while(i_grid1 < n_grid1) {

        tmpJ[i_grid1            ] = 0.0;
        tmpJ[i_grid1 +   n_grid1] = 0.0;
        tmpJ[i_grid1 + 2*n_grid1] = 0.0;

        wr1_tmp = wr1[i_grid1];

        ii_grid1 = i_grid1 + ie * nn;

        for(ie = 0; ie < ne_b; ie++) {

            tmpJ[i_grid1            ] += wr1_tmp * int2_grad1_u12[ii_grid1              ];
            tmpJ[i_grid1 +   n_grid1] += wr1_tmp * int2_grad1_u12[ii_grid1 +     n_grid1];
            tmpJ[i_grid1 + 2*n_grid1] += wr1_tmp * int2_grad1_u12[ii_grid1 + 2 * n_grid1];

        }

        i_grid1 += blockDim.x * gridDim.x;

    }

}



extern "C" void no_2e_tmpJ_cs(int n_grid1, int n_mo, int ne_b,
                              double * wr1, double * int2_grad1_u12,
                              double * tmpJ) {

    int nBlocks, blockSize;

    blockSize = 32;
    nBlocks = (n_grid1 + blockSize - 1) / blockSize;

    printf("lunching no_2e_tmpJ_cs_kernel with %d blocks and %d threads/block\n", nBlocks, blockSize);

    no_2e_tmpJ_cs_kernel<<<nBlocks, blockSize>>>(n_grid1, n_mo, ne_b,
                                                 wr1, int2_grad1_u12,
                                                 tmpJ);

}


